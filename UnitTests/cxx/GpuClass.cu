//
// Created by mbahassan on 2/21/25.
//

#include "GpuClass.cuh"
#include "GpuClassKernel.cuh"

#include "Tools/CudaHelper.hpp"

GpuClass::GpuClass(const Particle* particle, const int size):
size_(size)
{
    hostToDevice(particle, size_, &devParticle);
}


GpuClass::~GpuClass()
{
    hipFree(devParticle);
}

void GpuClass::printHellow() const
{
    kernel<<<1,10>>>(devParticle, size_);
}
