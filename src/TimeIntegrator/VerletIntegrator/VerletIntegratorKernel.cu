#include "hip/hip_runtime.h"
//
// Created by iqraa on 27-2-25.
//

#ifndef VERLET_INTEGRATOR_KERNEL_CUH
#define VERLET_INTEGRATOR_KERNEL_CUH

#include <>

#include "Particle/Spherical.h"
#include "Tools/ArthmiticOperator/MathOperators.hpp"

template<typename ParticleType>
__global__ void verletIntegratorKernel(ParticleType *particle, const int size_, const float dt)
{
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx >= size_) return;

  // Update position
  particle[idx].position += particle[idx].velocity * dt;

  // Update velocity
  // particle[idx].velocity = particle[idx].force / particle[idx].mass *dt;

  // Update Angular velocity
  // particle[idx].angularVel += particle[idx].torque / particle[idx].inertia * dt;
}

#endif //VERLET_INTEGRATOR_KERNEL_CUH
